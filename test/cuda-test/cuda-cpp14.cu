//  Direct NVCC command line example:
//
//  nvcc ./cuda-cpp14.cu -x cu -I"../include" -l"fmtd" -L"../build/Debug" \
//       -std=c++14 -Xcompiler /std:c++14 -Xcompiler /Zc:__cplusplus

// Ensure that we are using the latest C++ standard for NVCC
// The version is C++14
//
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#c-cplusplus-language-support
// https://en.cppreference.com/w/cpp/preprocessor/replace#Predefined_macros
static_assert(__cplusplus >= 201402L, "expect C++ 2014 for nvcc");

#include <fmt/base.h>

#include <hip/hip_runtime.h>
#include <iostream>

extern auto make_message_cpp() -> std::string;
extern auto make_message_cuda() -> std::string;

int main() {
  std::cout << make_message_cuda() << std::endl;
  std::cout << make_message_cpp() << std::endl;
}

auto make_message_cuda() -> std::string {
  return fmt::format("nvcc compiler \t: __cplusplus == {}", __cplusplus);
}
